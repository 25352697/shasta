#include "hip/hip_runtime.h"
#include <algorithm>
#include <stdint.h>
#include <thrust/sort.h>
#include <thrust/scan.h>
#include <mutex>
#include <vector>
#include <sys/time.h>
#include "GPU.h"


#define BAND_SIZE 256 
#define LOG_BLOCK_SIZE 8
#define LOG_NUM_BLOCKS 8
#define MAX_MARKER_OCC 8
#define HASH_PER_BLOCK MAX_MARKERS_PER_READ
#define MAX_MARKER_OCC_ELEM (1+MAX_MARKER_OCC)/2
#define BYTES_PER_HASH 4*(1+MAX_MARKER_OCC_ELEM)

#define BLOCK_SIZE (1 << LOG_BLOCK_SIZE)
#define NUM_BLOCKS (1 << LOG_NUM_BLOCKS)
#define INVALID_ID 0xffffffff

std::mutex* gpu_lock;

uint32_t** h_reads_pinned;

uint32_t** d_reads;
uint64_t** d_read_pairs;
uint32_t** d_alignments;

uint32_t** d_index_table;
uint32_t** d_hash;
uint32_t** d_marker_h;
uint32_t** d_tb_mem;
uint32_t** d_num_traceback;
uint32_t** d_common_markers;
uint32_t** d_num_common_markers;
uint32_t** d_num_hash_values;

__global__
void find_common_markers (int n, uint32_t* d_num_hash_values, uint32_t* d_reads, uint64_t* d_read_pairs, uint32_t* d_index_table, uint32_t* d_hash, uint32_t* d_marker_h, uint32_t* d_common_markers, uint32_t* d_num_common_markers) {
    int tx = threadIdx.x;
    int bs = blockDim.x;
    int bx = blockIdx.x;
    int gs = gridDim.x;

    //start address of hash table
    uint32_t hs = bx*HASH_PER_BLOCK*(BYTES_PER_HASH/4);
    uint32_t prev_rid1 = INVALID_ID;

    __shared__ uint32_t rid1;
    __shared__ uint32_t s1, e1, s2, e2;
    __shared__ uint32_t h1_arr[BLOCK_SIZE][MAX_MARKER_OCC];
    __shared__ uint32_t prefix[1+BLOCK_SIZE];
    __shared__ uint32_t curr_offset[BLOCK_SIZE];
    __shared__ int has_failed;

    int chunk_size = (n/gs);
    chunk_size += 1;

    int si = bx*chunk_size;
    int ei = si + chunk_size;
    if (ei > n) {
        ei = n;
    }
    
    for (int i = tx; i < HASH_PER_BLOCK; i+=bs) {
        d_hash[hs+i*(1+MAX_MARKER_OCC_ELEM)] = INVALID_ID;
    }

    __syncthreads();

    for (int i = si; i < ei; i++) {
        __syncthreads();
        if (tx == 0) {
            uint64_t v1 = d_read_pairs[2*i];
            uint64_t v2 = d_read_pairs[2*i+1];
            
            s1 = (v1 >> 32);
            e1 = s1 + ((v1 << 32) >> 32);

            s2 = (v2 >> 32);
            e2 = s2 + ((v2 << 32) >> 32);
            
            rid1 = s1;

            prefix[0] = bx*HASH_PER_BLOCK;

            has_failed = 0;
            if ((s1 == e1) || (s2 == e2)) {
                has_failed = 1;
            }
        }

        curr_offset[tx] = 0;
        prefix[tx+1] = 0;

        __syncthreads();

        if (has_failed > 0) {
            if (tx == 0) {
                d_num_common_markers[i] = 0;
            }
            continue;
        }

        if (rid1 != prev_rid1) {
            prev_rid1 = rid1;
            for (uint32_t p = 0; p < e1-s1; p += bs) {
                if (s1+p+tx < e1) {
                    uint32_t idx;
                    uint32_t k, h1;

                    k = d_reads[s1+p+tx];
                    
                    h1 = (~k) + (k << 11);
                    h1 = h1 + (k >> 12);
                    h1 = (h1 + (h1 << 3)) + (h1 << 8);

                    idx = h1 % bs;

                    atomicAdd(&prefix[idx+1], 1);
                }
            }

            __syncthreads();

            if (tx == 0) {
                for (int r = 0; r < BLOCK_SIZE; r++) {
                    prefix[1+r] += prefix[r];
                }
            }

            curr_offset[tx] = 0; 
            __syncthreads();

            for (int p = 0; p < e1-s1; p += bs) {
                if (s1+p+tx < e1) {
                    uint32_t idx;
                    uint32_t k, h1;

                    k = d_reads[s1+p+tx];
                    
                    h1 = (~k) + (k << 11);
                    h1 = h1 + (k >> 12);
                    h1 = (h1 + (h1 << 3)) + (h1 << 8);
                    
                    idx = h1 % bs;

                    uint32_t hst = prefix[idx];
                    uint16_t off = atomicAdd(&curr_offset[idx], 1);

                    uint32_t val = h1 % HASH_PER_BLOCK;
                    val = (val << 16) + (p+tx);

                    d_marker_h[hst+off] = val;
                }
            }

            __syncthreads();

            uint32_t mhs = prefix[tx];
            uint32_t mhe = prefix[tx+1];

            for (uint32_t m = mhs; m < mhe; m++) {
                uint32_t v = d_marker_h[m];
                uint32_t h = (v >> 16);
                uint32_t p = ((v << 16) >> 16);

                uint32_t hidx = hs + h*(1+MAX_MARKER_OCC_ELEM);
                uint32_t rid = d_hash[hidx];

                if (rid != rid1) {
                    d_hash[hidx] = rid1;
                    d_hash[hidx+1] = p+1;
                }
                else {
                    for (int q = 1; q <= MAX_MARKER_OCC_ELEM; q++) {
                        uint32_t val = d_hash[hidx+q];
                        uint32_t l, u;
                        l = ((val << 16) >> 16);
                        u = (val >> 16);
                        if (l == 0) {
                            d_hash[hidx+q] = (p+1);
                            break;
                        }
                        else if (u == 0) {
                            d_hash[hidx+q] = val + ((p+1) << 16);
                            if (q < MAX_MARKER_OCC_ELEM) {
                                d_hash[hidx+q+1] = 0; 
                            }
                            break;
                        }
                    }
                }
            }
        }
        
        __syncthreads();
        
        if (has_failed > 0) {
            if (tx == 0) {
                d_num_common_markers[i] = 0;
            }
            continue;
        }

        if (tx == 0) {
            for (int r = 0; r < bs-1; r++) {
                curr_offset[1+r] += curr_offset[r];
            }
            prefix[0] = i*HASH_PER_BLOCK;
        }

        prefix[1+tx] = 0;
        __syncthreads();

        int n1 = 0;

        for (uint32_t p = 0; p < e2-s2; p += bs) {
            uint32_t h1, rid;
            uint32_t hidx;
            uint32_t l, u;
            uint32_t k;

            rid = INVALID_ID;

            if (s2+p+tx < e2) {
                k = d_reads[s2+p+tx];
                
                h1 = (~k) + (k << 11);
                h1 = h1 + (k >> 12);
                h1 = (h1 + (h1 << 3)) + (h1 << 8);


                h1 = h1 % HASH_PER_BLOCK;

                hidx = hs + (1+MAX_MARKER_OCC_ELEM)*h1;

                rid = d_hash[hidx];
            }

            n1 = 0;
            if (rid == rid1) {
                for (int q = 0; q < MAX_MARKER_OCC_ELEM; q++) {
                    uint32_t val = d_hash[hidx+q+1];
                    l = ((val << 16) >> 16);
                    u = (val >> 16);

                    if (l != 0) {
                        if (k == d_reads[s1+l-1]) {
                            h1_arr[tx][n1] = (p+tx+1) + (l << 16);
                            n1++;
                        }
                        if (u != 0) {
                            if (k == d_reads[s1+u-1]) {
                                h1_arr[tx][n1] = (p+tx+1) + (u << 16);
                                n1++;
                            }
                        }
                        else {
                            break;
                        }
                    }
                    else {
                        break;
                    }
                }
            }

            prefix[1+tx] = n1;
            __syncthreads();

            if (tx == 0) {
                for (int d = 0; d < BLOCK_SIZE; d++) {
                    prefix[1+d] += prefix[d];
                }
            }
            __syncthreads();

            uint32_t addr_s = prefix[tx];
            uint32_t addr_e = prefix[1+tx];

            for (uint32_t addr = addr_s; addr < addr_e; addr++) {
                if (addr-i*HASH_PER_BLOCK < HASH_PER_BLOCK) {
                    d_common_markers[addr] = h1_arr[tx][addr-addr_s];
                }
            }

            __syncthreads();
            
            if (tx == 0) {
                uint32_t addr = prefix[BLOCK_SIZE];
                prefix[0] = addr;
                if (addr-i*HASH_PER_BLOCK < HASH_PER_BLOCK) {
                    d_common_markers[addr] = 0;
                }
            }
            __syncthreads();
        }
        
        __syncthreads();

        int num_common_markers =  prefix[BLOCK_SIZE] - i*HASH_PER_BLOCK;

        if (tx == 0) {
            // TODO: fail if >= HASH_PER_BLOCK?
            d_num_common_markers[i] = (num_common_markers < HASH_PER_BLOCK) ? num_common_markers : HASH_PER_BLOCK;
        }

        __syncthreads();
    }
}

__global__
void find_traceback (int n, size_t maxSkip, uint32_t* d_marker_h, uint32_t* d_common_markers, uint32_t* d_num_common_markers, uint32_t* d_tb_mem, uint32_t* d_alignments, uint32_t* d_num_traceback) {
    int tx = threadIdx.x;
    int bs = blockDim.x;
    int bx = blockIdx.x;
    int gs = gridDim.x;

    __shared__ uint32_t score[BAND_SIZE];
    __shared__ uint32_t score_pos[BAND_SIZE];
    __shared__ int num_common_markers;
    __shared__ bool stop_shared;

    for (int i = bx; i < n; i += gs) {
        uint32_t max_score = 0, max_score_pos = 0;
        uint32_t addr1 = i*MAX_MARKERS_PER_READ;
        uint32_t addr2 = bx*HASH_PER_BLOCK;

//        uint32_t start_addr = 0;
//        if (i > 0) {
//            start_addr = d_num_traceback[i-1];
//        }
//        uint32_t num_tb = d_num_traceback[i] - start_addr;
//        
        if (tx == 0) {
            num_common_markers = d_num_common_markers[i];
//            d_alignments[start_addr] = 0;
            d_alignments[addr1] = 0;
        }
        score[tx] = 0;

        __syncthreads();

        for (int p = 0; p < num_common_markers; p++) {
            uint32_t v = d_common_markers[addr1+p];
            uint32_t l = ((v << 16) >> 16);
            uint32_t u = (v >> 16);

            int ptr = p - tx - 1;

            score[tx] = 1;
            score_pos[tx] = p;

            bool stop = false;
            __syncthreads();

            while (!stop) {
                uint32_t l1, u1;
                if (ptr >= 0) {
                    uint32_t v1 = d_common_markers[addr1+ptr];
                    l1 = ((v1 << 16) >> 16);
                    u1 = (v1 >> 16);
                    if ((l1 < l) && (u1 < u) && (u-u1 < 8) && (l-l1 < 8)) {
                        uint32_t pscore = d_marker_h[addr2+ptr];
                        if (score[tx] < pscore+1) { 
                            score[tx] = pscore+1;
                            score_pos[tx] = ptr;
                        }
                    }
                }
                ptr -= bs;
                if (tx == 0) {
                    if ((ptr < 0) || (l-l1 < 8))  {
                        stop_shared = true;
                    }
                    else {
                        stop_shared = false;
                    }
                }
                __syncthreads();
                stop = stop_shared;
            }

            __syncthreads();

            // parallel reduction (max)
            for(unsigned int s = 1; s < bs; s *= 2) {
                if (tx % (2*s) == 0) {
                    if (score[tx] < score[tx+s]) { 
                        score[tx] = score[tx + s];
                        score_pos[tx] = score_pos[tx + s];
                    }
                }
                __syncthreads();
            }
            
            if (tx == 0) {
                d_marker_h[addr2+p] = score[0];
                d_tb_mem[addr2+p] = score_pos[0];
                if (score[0] > max_score) {
                    max_score = score[0];
                    max_score_pos = score_pos[0];
                }
            }
            __syncthreads();
        }

        __syncthreads();

        if (tx == 0) {
            int num_ptr = 0;

            if (max_score > 0) {
                int curr_pos = max_score_pos;
                int prev_pos = max_score_pos + 1;

                while ((curr_pos >= 0) && (prev_pos > curr_pos)) {
                    prev_pos = curr_pos;
                    d_alignments[addr1+num_ptr] = d_common_markers[addr1+curr_pos];
                    num_ptr++;
                    curr_pos = d_tb_mem[addr2+curr_pos];
                }
            }

            if (num_ptr < HASH_PER_BLOCK) {
                d_alignments[addr1+num_ptr] = 0;
            }
//            d_num_traceback[i] = num_ptr;
        }
        __syncthreads();
    }
}

extern "C" int initializeProcessors () {
    int nDevices;

    hipGetDeviceCount(&nDevices);
    //    for (int i = 0; i < nDevices; i++) {
    //        hipDeviceProp_t prop;
    //        hipGetDeviceProperties(&prop, i);
    //        printf("Device Number: %d\n", i);
    //        printf("  Device name: %s\n", prop.name);
    //        printf("  Memory Clock Rate (KHz): %d\n",
    //                prop.memoryClockRate);
    //        printf("  Memory Bus Width (bits): %d\n",
    //                prop.memoryBusWidth);
    //        printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
    //                2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    //    }

    gpu_lock = new std::mutex[nDevices];
    
    h_reads_pinned = (uint32_t**) malloc(nDevices*sizeof(uint32_t*));

    d_reads = (uint32_t**) malloc(nDevices*sizeof(uint32_t*));
    d_read_pairs = (uint64_t**) malloc(nDevices*sizeof(uint64_t*));
    d_alignments = (uint32_t**) malloc(nDevices*sizeof(uint32_t*));

    d_index_table = (uint32_t**) malloc(nDevices*sizeof(uint32_t*));
    d_hash = (uint32_t**) malloc(nDevices*sizeof(uint32_t*));
    d_marker_h = (uint32_t**) malloc(nDevices*sizeof(uint32_t*));
    d_tb_mem = (uint32_t**) malloc(nDevices*sizeof(uint32_t*));
    d_num_traceback = (uint32_t**) malloc(nDevices*sizeof(uint32_t*));
    d_common_markers = (uint32_t**) malloc(nDevices*sizeof(uint32_t*));
    d_num_common_markers = (uint32_t**) malloc(nDevices*sizeof(uint32_t*));
    d_num_hash_values = (uint32_t**) malloc(nDevices*sizeof(uint32_t*));

    hipError_t err;
    size_t num_bytes;

    for (int k=0; k<nDevices; k++) {
        gpu_lock[k].lock();

        err = hipSetDevice(k);
        if (err != hipSuccess) {
            fprintf(stderr, "GPU_ERROR: could not set device %d!\n", k);
            exit(1);
        }
        
        num_bytes = 2*GPU_BATCH_SIZE*MAX_MARKERS_PER_READ*sizeof(uint32_t);
        err = hipHostMalloc(&h_reads_pinned[k], num_bytes); 
        if (err != hipSuccess) {
            fprintf(stderr, "ERROR: hipHostMalloc failed!\n");
            exit(1);
        }

        num_bytes = 2*GPU_BATCH_SIZE*MAX_MARKERS_PER_READ*sizeof(uint32_t);
        if (k==0)
            fprintf(stdout, "\t-Requesting %3.0e bytes on GPU\n", (double)num_bytes);
        err = hipMalloc(&d_reads[k], num_bytes); 
        if (err != hipSuccess) {
            fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
            exit(1);
        }
        
        num_bytes = 2*GPU_BATCH_SIZE*sizeof(uint64_t);
        if (k==0)
            fprintf(stdout, "\t-Requesting %3.0e bytes on GPU\n", (double)num_bytes);
        err = hipMalloc(&d_read_pairs[k], num_bytes); 
        if (err != hipSuccess) {
            fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
            exit(1);
        }
        
        num_bytes = GPU_BATCH_SIZE*MAX_MARKERS_PER_READ*sizeof(uint32_t);
        if (k==0)
            fprintf(stdout, "\t-Requesting %3.0e bytes on GPU\n", (double)num_bytes);
        err = hipMalloc(&d_alignments[k], num_bytes); 
        if (err != hipSuccess) {
            fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
            exit(1);
        }

        num_bytes = NUM_BLOCKS*MAX_MARKERS_PER_READ*sizeof(uint32_t);
        if (k==0)
            fprintf(stdout, "\t-Requesting %3.0e bytes on GPU\n", (double)num_bytes);
        err = hipMalloc(&d_index_table[k], num_bytes); 
        if (err != hipSuccess) {
            fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
            exit(1);
        }
        
        num_bytes = NUM_BLOCKS*HASH_PER_BLOCK*BYTES_PER_HASH;
        if (k==0)
            fprintf(stdout, "\t-Requesting %3.0e bytes on GPU\n", (double)num_bytes);
        err = hipMalloc(&d_hash[k], num_bytes); 
        if (err != hipSuccess) {
            fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
            exit(1);
        }
        
        num_bytes = NUM_BLOCKS*HASH_PER_BLOCK*sizeof(uint32_t);
        if (k==0)
            fprintf(stdout, "\t-Requesting %3.0e bytes on GPU\n", (double)num_bytes);
        err = hipMalloc(&d_marker_h[k], num_bytes); 
        if (err != hipSuccess) {
            fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
            exit(1);
        }
        
        num_bytes = NUM_BLOCKS*HASH_PER_BLOCK*sizeof(uint32_t);
        if (k==0)
            fprintf(stdout, "\t-Requesting %3.0e bytes on GPU\n", (double)num_bytes);
        err = hipMalloc(&d_tb_mem[k], num_bytes); 
        if (err != hipSuccess) {
            fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
            exit(1);
        }
        
        num_bytes = GPU_BATCH_SIZE*sizeof(uint32_t);
        if (k==0)
            fprintf(stdout, "\t-Requesting %3.0e bytes on GPU\n", (double)num_bytes);
        err = hipMalloc(&d_num_traceback[k], num_bytes); 
        if (err != hipSuccess) {
            fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
            exit(1);
        }
        
//        num_bytes = GPU_BATCH_SIZE*sizeof(uint32_t);
//        if (k==0)
//            fprintf(stdout, "\t-Requesting %3.0e bytes on GPU\n", (double)num_bytes);
//        err = hipMalloc(&d_prefix_num_traceback[k], num_bytes); 
//        if (err != hipSuccess) {
//            fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
//            exit(1);
//        }
        
        num_bytes = GPU_BATCH_SIZE*sizeof(uint32_t);
        if (k==0)
            fprintf(stdout, "\t-Requesting %3.0e bytes on GPU\n", (double)num_bytes);
        err = hipMalloc(&d_num_common_markers[k], num_bytes); 
        if (err != hipSuccess) {
            fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
            exit(1);
        }
        
        num_bytes = GPU_BATCH_SIZE*HASH_PER_BLOCK*sizeof(uint32_t);
        if (k==0)
            fprintf(stdout, "\t-Requesting %3.0e bytes on GPU\n", (double)num_bytes);
        err = hipMalloc(&d_common_markers[k], num_bytes); 
        if (err != hipSuccess) {
            fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
            exit(1);
        }
        
        num_bytes = sizeof(uint32_t);
        if (k==0)
            fprintf(stdout, "\t-Requesting %3.0e bytes on GPU\n", (double)num_bytes);
        err = hipMalloc(&d_num_hash_values[k], num_bytes); 
        if (err != hipSuccess) {
            fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
            exit(1);
        }
        err = hipMemset(d_num_hash_values[k], 0x0, num_bytes); 
        if (err != hipSuccess) {
            fprintf(stderr, "GPU_ERROR: hipMemset failed!\n");
            exit(1);
        }
        
        gpu_lock[k].unlock();
    }

    return nDevices;
}

void alignBatchGPU (size_t n, size_t deviceId, size_t maxSkip, size_t num_pos, uint32_t* h_reads, uint64_t* h_read_pairs, uint32_t* h_alignments) {
    hipError_t err;

    bool report_time = false;

    size_t k = deviceId;

    struct timeval t1, t2, t3;
    long useconds, seconds, mseconds;

    std::memcpy(h_reads_pinned[k], h_reads, num_pos*sizeof(uint32_t));

    uint32_t* h_prefix_num_tb = (uint32_t*) malloc(n*sizeof(uint32_t));

    gpu_lock[k].lock();
    
    gettimeofday(&t1, NULL);

    err = hipSetDevice(k);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: could not set device %zu!\n", k);
        exit(1);
    }

    err = hipMemcpy(d_reads[k], h_reads_pinned[k], num_pos*sizeof(uint32_t), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: hipMemcpy failed!\n");
        exit(1);
    }
    
    err = hipMemcpy(d_read_pairs[k], h_read_pairs, 2*n*sizeof(uint64_t), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: hipMemcpy failed!\n");
        exit(1);
    }

    find_common_markers <<<NUM_BLOCKS, BLOCK_SIZE>>> (n, d_num_hash_values[k], d_reads[k], d_read_pairs[k], d_index_table[k], d_hash[k], d_marker_h[k], d_common_markers[k], d_num_common_markers[k]);

    err = hipMemcpy(h_prefix_num_tb, d_num_common_markers[k], n*sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: hipMemcpy failed!\n");
        exit(1);
    }

    thrust::inclusive_scan(h_prefix_num_tb, h_prefix_num_tb+n, h_prefix_num_tb);
    
    err = hipMemcpy(d_num_traceback[k], h_prefix_num_tb, n*sizeof(uint32_t), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: hipMemcpy failed!\n");
        exit(1);
    }

    err = hipMemcpy(h_prefix_num_tb, d_common_markers[k], 320*sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: hipMemcpy failed!\n");
        exit(1);
    }
    
//    for (uint32_t i=0; i<320; i++) {
//        uint32_t v, l, u;
//        v = h_prefix_num_tb[i];
//        u = (v >> 16);
//        l = ((v << 16) >> 16);
//        fprintf(stderr, "%u: %u %u\n", i, u, l);
////        fprintf(stderr, "%u: %u\n", i, v);
//    }

    find_traceback <<<NUM_BLOCKS, BAND_SIZE>>> (n, maxSkip, d_marker_h[k], d_common_markers[k], d_num_common_markers[k], d_tb_mem[k], d_alignments[k], d_num_traceback[k]);

    gettimeofday(&t2, NULL);

    uint32_t total_num_tb = 0;
    if (n > 0) {
        total_num_tb = h_prefix_num_tb[n-1];
    }

    uint32_t* h_tb;
    err =  hipHostMalloc ((uint32_t**) &h_tb, total_num_tb*sizeof(uint32_t));

    //    err = hipMemcpy(h_tb, d_alignments[k], total_num_tb*sizeof(uint32_t), hipMemcpyDeviceToHost);
    //    if (err != hipSuccess) {
    //        fprintf(stderr, "GPU_ERROR: hipMemcpy failed!\n");
    //        exit(1);
    //    }

    gpu_lock[k].unlock();

    gettimeofday(&t3, NULL);

    for (int i=0; i<n; i++) {
        uint32_t start;
        //uint32_t end;
        uint32_t num_tb;
        start = 0;
        if (i > 0) {
            start = h_prefix_num_tb[i-1];
        }
        //end = h_prefix_num_tb[i];
        //TODO: remove after fixind find_traceback
        num_tb = 0; //end - start;
        std::memcpy(&h_alignments[i*MAX_MARKERS_PER_READ], &h_tb[start], num_tb*sizeof(uint32_t));
        if (num_tb < MAX_MARKERS_PER_READ) {
            h_alignments[i*MAX_MARKERS_PER_READ+num_tb] = 0; 
        }
    }

    if (report_time) {
        useconds = t2.tv_usec - t1.tv_usec;
        seconds = t2.tv_sec - t1.tv_sec;
        mseconds = ((seconds) * 1000 + useconds/1000.0) + 0.5;

        fprintf(stderr, "Time elapsed (t2-t1): %ld msec \n", mseconds);

        useconds = t3.tv_usec - t1.tv_usec;
        seconds = t3.tv_sec - t1.tv_sec;
        mseconds = ((seconds) * 1000 + useconds/1000.0) + 0.5;

        fprintf(stderr, "Time elapsed (t3-t1): %ld msec \n", mseconds);
        
        fprintf(stderr, "Num bytes transferred (reads): %zu \n", num_pos*4);
        fprintf(stderr, "Num bytes transferred (traceback): %u \n", total_num_tb*4);
    }

    hipFree(h_tb);
    free(h_prefix_num_tb);

    return;
}

extern "C" void shutdownProcessors(int nDevices) {
    for (int k=0; k<nDevices; k++) {
        hipFree(h_reads_pinned[k]);

        hipFree(d_reads[k]);
        hipFree(d_read_pairs[k]);
        hipFree(d_alignments[k]);

        hipFree(d_index_table[k]);
        hipFree(d_hash[k]);
        hipFree(d_marker_h[k]);
        hipFree(d_tb_mem[k]);
        hipFree(d_num_traceback[k]);
        hipFree(d_common_markers[k]);
        hipFree(d_num_common_markers[k]);
        hipFree(d_num_hash_values[k]);
    }
        
    free(h_reads_pinned);
    
    free(d_reads);
    free(d_read_pairs);
    free(d_alignments);

    free(d_index_table);
    free(d_hash);
    free(d_marker_h);
    free(d_tb_mem);
    free(d_num_traceback);
    free(d_common_markers);
    free(d_num_common_markers);
    free(d_num_hash_values);

    delete(gpu_lock);

}
