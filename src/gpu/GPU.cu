#include "hip/hip_runtime.h"
#include <algorithm>
#include <stdint.h>
#include <thrust/sort.h>
#include <thrust/scan.h>
#include <thrust/binary_search.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <mutex>
#include <vector>
#include <sys/time.h>
#include "GPU.h"


#define BAND_SIZE 32 
#define LOG_BLOCK_SIZE 7
#define LOG_NUM_BLOCKS 10

#define BLOCK_SIZE (1 << LOG_BLOCK_SIZE)
#define NUM_BLOCKS (1 << LOG_NUM_BLOCKS)

std::mutex* gpu_lock;

uint32_t num_unique_markers;

uint32_t** d_alignments;
uint32_t** d_marker_h;
uint32_t** d_tb_mem;
uint32_t** d_num_traceback;
uint32_t** d_common_markers;
uint32_t** d_num_common_markers;

__global__
void find_common_markers (uint64_t maxMarkerFrequency, uint64_t n, uint32_t num_unique_markers, uint64_t* read_pairs, uint64_t* index_table, uint64_t* rid_marker_pos, uint64_t* sorted_rid_marker_pos, uint32_t* num_common_markers, uint32_t* common_markers)
{
    int tx = threadIdx.x;
    int bs = blockDim.x;
    int bx = blockIdx.x;
    int gs = gridDim.x;

    uint64_t m_mask = ((uint64_t) 1 << 32) - 1;
    uint64_t p_mask = ((uint64_t) 1 << SHASTA_LOG_MAX_MARKERS_PER_READ) - 1;
    
    __shared__ uint32_t prefix[1+BLOCK_SIZE];

    __syncthreads();

    for (int i = bx; i < n; i+=gs) {
        if (tx == 0) {
            prefix[tx] = i*SHASTA_MAX_MARKERS_PER_READ;
            num_common_markers[i] = 0;
        }
        __syncthreads();

        uint64_t v1 = read_pairs[2*i];
        uint64_t v2 = read_pairs[2*i+1];
        uint64_t rid1 = (v1 >> 32);
        uint64_t rid2 = (v2 >> 32);
        uint64_t l1 = ((v1 << 32) >> 32);
        uint64_t l2 = ((v2 << 32) >> 32);

        if ((l1 > 0) && (l2 > 0)) {
            uint64_t s2 = index_table[rid2*num_unique_markers];
            uint64_t e2 = index_table[(rid2+1)*num_unique_markers];

            for (uint64_t j = s2; j < e2; j += bs) {
                uint64_t idx = tx+j;
                uint64_t marker;
                uint64_t sm1=0, sm2=0, em1=0, em2=0;

                prefix[1+tx] = 0; 

                if (idx < e2) {
                    uint64_t v = rid_marker_pos[idx];
                    marker = ((v >> SHASTA_LOG_MAX_MARKERS_PER_READ) & m_mask);

                    sm1 = index_table[rid1*num_unique_markers+marker];
                    em1 = index_table[rid1*num_unique_markers+marker+1];
                    sm2 = index_table[rid2*num_unique_markers+marker];
                    em2 = index_table[rid2*num_unique_markers+marker+1];

                    if ((em1 - sm1 <= maxMarkerFrequency) && (em2 - sm2 <= maxMarkerFrequency)) {
                        prefix[1+tx] = (em1-sm1);
                    }
                }

                __syncthreads();

                if (tx == 0) {
                    for (int r = 0; r < BLOCK_SIZE; r++) {
                        prefix[1+r] += prefix[r];
                    }
                }

                __syncthreads();

                uint32_t mhs = prefix[tx];
                uint32_t mhe = prefix[1+tx];

                for (uint64_t k1 = 0; k1 < (mhe-mhs); k1++) {
                    if (mhs+k1 < (i+1)*SHASTA_MAX_MARKERS_PER_READ) {
                        uint64_t sv1 = sorted_rid_marker_pos[sm1+k1];
                        uint32_t cm = (sv1 & p_mask) + 1;
                        cm = (cm << 16) + (1+idx-s2);
                        common_markers[mhs+k1] = cm;
                    }
                }

                __syncthreads();

                if (tx == 0) {
                    prefix[tx] = prefix[BLOCK_SIZE];
                }

                __syncthreads();
            }

            if (tx == 0) {
                uint32_t num_common = prefix[tx] - i*SHASTA_MAX_MARKERS_PER_READ;
                if (num_common < SHASTA_MAX_MARKERS_PER_READ) {
                    num_common_markers[i] = num_common;
                }
            }
        }

        __syncthreads();
    }
}

__global__
void find_traceback (int n, size_t maxSkip, uint32_t* d_marker_h, uint32_t* d_common_markers, uint32_t* d_num_common_markers, uint32_t* d_tb_mem, uint32_t* d_alignments, uint32_t* d_num_traceback) {
    int tx = threadIdx.x;
    int bs = blockDim.x;
    int bx = blockIdx.x;
    int gs = gridDim.x;

    __shared__ uint32_t score[BAND_SIZE];
    __shared__ uint32_t score_pos[BAND_SIZE];
    __shared__ int num_common_markers;
    __shared__ bool stop_shared;

    for (int i = bx; i < n; i += gs) {
        uint32_t max_score = 0, max_score_pos = 0;
        uint32_t addr1 = i*SHASTA_MAX_MARKERS_PER_READ;
        uint32_t addr2 = bx*SHASTA_MAX_MARKERS_PER_READ;
        uint32_t addr3 = i*SHASTA_MAX_TB;

        if (tx == 0) {
            num_common_markers = d_num_common_markers[i];
            d_alignments[addr3] = 0;
        }
        score[tx] = 0;

        __syncthreads();

        for (int p = 0; p < num_common_markers; p++) {
            uint32_t v = d_common_markers[addr1+p];
            uint32_t l = ((v << 16) >> 16);
            uint32_t u = (v >> 16);

            int ptr = p - tx - 1;

            score[tx] = 1;
            score_pos[tx] = p;

            bool stop = false;
            __syncthreads();

            while (!stop) {
                uint32_t l1, u1;
                if (ptr >= 0) {
                    uint32_t v1 = d_common_markers[addr1+ptr];
                    l1 = ((v1 << 16) >> 16);
                    u1 = (v1 >> 16);
                    if ((l1 < l) && (u1 < u) && (u-u1 < maxSkip) && (l-l1 < maxSkip)) {
                        uint32_t pscore = d_marker_h[addr2+ptr];
                        if (score[tx] < pscore+1) { 
                            score[tx] = pscore+1;
                            score_pos[tx] = ptr;
                        }
                    }
                }
                ptr -= bs;
                if (tx == bs-1) {
                    if ((ptr < 0) || (l-l1 >= maxSkip))  {
                        stop_shared = true;
                    }
                    else {
                        stop_shared = false;
                    }
                }
                __syncthreads();
                stop = stop_shared;
            }

            __syncthreads();

            // parallel reduction (max)
            for(unsigned int s = 1; s < bs; s *= 2) {
                if (tx % (2*s) == 0) {
                    if (score[tx] < score[tx+s]) { 
                        score[tx] = score[tx + s];
                        score_pos[tx] = score_pos[tx + s];
                    }
                }
                __syncthreads();
            }
            
            if (tx == 0) {
                d_marker_h[addr2+p] = score[0];
                d_tb_mem[addr2+p] = score_pos[0];
                if (score[0] > max_score) {
                    max_score = score[0];
                    max_score_pos = score_pos[0];
                }
            }
            __syncthreads();
        }

        __syncthreads();

        if (tx == 0) {
            int num_ptr = 0;

            if (max_score > 0) {
                int curr_pos = max_score_pos;
                int prev_pos = max_score_pos + 1;

                while ((curr_pos >= 0) && (prev_pos > curr_pos)) {
                    prev_pos = curr_pos;
                    if (num_ptr < SHASTA_MAX_TB) {
                        d_alignments[addr3+num_ptr] = d_common_markers[addr1+curr_pos];
                    }
                    num_ptr++;
                    curr_pos = d_tb_mem[addr2+curr_pos];
                }
            }

            if (num_ptr < SHASTA_MAX_TB) {
                d_alignments[addr3+num_ptr] = 0;
            }
            else {
                d_alignments[addr3] = 0;
            }
            d_num_traceback[i] = num_ptr;
        }
        __syncthreads();
    }
}

extern "C" int shasta_initializeProcessors (size_t numUniqueMarkers) {
    int nDevices;

    num_unique_markers = (uint32_t) numUniqueMarkers;

    hipGetDeviceCount(&nDevices);
    //    for (int i = 0; i < nDevices; i++) {
    //        hipDeviceProp_t prop;
    //        hipGetDeviceProperties(&prop, i);
    //        printf("Device Number: %d\n", i);
    //        printf("  Device name: %s\n", prop.name);
    //        printf("  Memory Clock Rate (KHz): %d\n",
    //                prop.memoryClockRate);
    //        printf("  Memory Bus Width (bits): %d\n",
    //                prop.memoryBusWidth);
    //        printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
    //                2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    //    }

    gpu_lock = new std::mutex[nDevices];
    
    d_alignments = (uint32_t**) malloc(nDevices*sizeof(uint32_t*));

    d_marker_h = (uint32_t**) malloc(nDevices*sizeof(uint32_t*));
    d_tb_mem = (uint32_t**) malloc(nDevices*sizeof(uint32_t*));
    d_num_traceback = (uint32_t**) malloc(nDevices*sizeof(uint32_t*));
    d_common_markers = (uint32_t**) malloc(nDevices*sizeof(uint32_t*));
    d_num_common_markers = (uint32_t**) malloc(nDevices*sizeof(uint32_t*));

    hipError_t err;
    size_t num_bytes;

    for (int k=0; k<nDevices; k++) {
        gpu_lock[k].lock();

        err = hipSetDevice(k);
        if (err != hipSuccess) {
            fprintf(stderr, "GPU_ERROR: could not set device %d!\n", k);
            exit(1);
        }
        
        num_bytes = SHASTA_GPU_BATCH_SIZE*SHASTA_MAX_TB*sizeof(uint32_t);
        if (k==0)
            fprintf(stdout, "\t-Requesting %3.0e bytes on GPU\n", (double)num_bytes);
        err = hipMalloc(&d_alignments[k], num_bytes); 
        if (err != hipSuccess) {
            fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
            exit(1);
        }

        num_bytes = NUM_BLOCKS*SHASTA_MAX_MARKERS_PER_READ*sizeof(uint32_t);
        if (k==0)
            fprintf(stdout, "\t-Requesting %3.0e bytes on GPU\n", (double)num_bytes);
        err = hipMalloc(&d_marker_h[k], num_bytes); 
        if (err != hipSuccess) {
            fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
            exit(1);
        }
        
        num_bytes = NUM_BLOCKS*SHASTA_MAX_MARKERS_PER_READ*sizeof(uint32_t);
        if (k==0)
            fprintf(stdout, "\t-Requesting %3.0e bytes on GPU\n", (double)num_bytes);
        err = hipMalloc(&d_tb_mem[k], num_bytes); 
        if (err != hipSuccess) {
            fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
            exit(1);
        }
        
        num_bytes = SHASTA_GPU_BATCH_SIZE*sizeof(uint32_t);
        if (k==0)
            fprintf(stdout, "\t-Requesting %3.0e bytes on GPU\n", (double)num_bytes);
        err = hipMalloc(&d_num_traceback[k], num_bytes); 
        if (err != hipSuccess) {
            fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
            exit(1);
        }
        
        num_bytes = SHASTA_GPU_BATCH_SIZE*sizeof(uint32_t);
        if (k==0)
            fprintf(stdout, "\t-Requesting %3.0e bytes on GPU\n", (double)num_bytes);
        err = hipMalloc(&d_num_common_markers[k], num_bytes); 
        if (err != hipSuccess) {
            fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
            exit(1);
        }
        
        num_bytes = SHASTA_GPU_BATCH_SIZE*SHASTA_MAX_MARKERS_PER_READ*sizeof(uint32_t);
        if (k==0)
            fprintf(stdout, "\t-Requesting %3.0e bytes on GPU\n", (double)num_bytes);
        err = hipMalloc(&d_common_markers[k], num_bytes); 
        if (err != hipSuccess) {
            fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
            exit(1);
        }
        
        gpu_lock[k].unlock();
    }

    return nDevices;
}

extern "C" void shasta_alignBatchGPU (size_t deviceId, size_t maxMarkerFrequency, size_t maxSkip, size_t n, uint32_t num_pos, uint32_t num_reads, uint64_t* batch_rid_marker_pos, uint64_t* batch_rid_markers, uint64_t* batch_read_pairs, uint32_t* h_alignments, uint32_t* h_num_traceback) {
    bool report_time = false;

    size_t k = deviceId;

    struct timeval t1, t2, t3;
    long useconds, seconds, mseconds;

    gpu_lock[k].lock();
    
    hipError_t err; 

    err = hipSetDevice(k);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: could not set device %zu!\n", k);
        exit(1);
    }
    
    gettimeofday(&t1, NULL);

    thrust::device_vector<uint64_t> t_d_rid_marker_pos (batch_rid_marker_pos, batch_rid_marker_pos + num_pos);
    thrust::device_vector<uint64_t> t_d_sorted_rid_marker_pos (batch_rid_marker_pos, batch_rid_marker_pos+num_pos);
    thrust::device_vector<uint64_t> t_d_rid_markers (batch_rid_markers, batch_rid_markers + num_reads*num_unique_markers+1);
    thrust::device_vector<uint64_t> t_d_read_pairs (batch_read_pairs, batch_read_pairs+2*n);
    thrust::device_vector<uint64_t> t_d_index_table (num_reads*num_unique_markers+1);

    thrust::sort(t_d_sorted_rid_marker_pos.begin(), t_d_sorted_rid_marker_pos.end());

    gettimeofday(&t2, NULL);

    thrust::lower_bound(t_d_sorted_rid_marker_pos.begin(),
            t_d_sorted_rid_marker_pos.end(),
            t_d_rid_markers.begin(),
            t_d_rid_markers.end(),
            t_d_index_table.begin());

    uint64_t* d_sorted_rid_marker_pos = thrust::raw_pointer_cast (t_d_sorted_rid_marker_pos.data());
    uint64_t* d_rid_marker_pos = thrust::raw_pointer_cast (t_d_rid_marker_pos.data());
    uint64_t* d_index_table = thrust::raw_pointer_cast (t_d_index_table.data());
    uint64_t* d_read_pairs = thrust::raw_pointer_cast (t_d_read_pairs.data());
    
    find_common_markers <<<NUM_BLOCKS, BLOCK_SIZE>>> (maxMarkerFrequency, n, num_unique_markers, d_read_pairs, d_index_table, d_rid_marker_pos, d_sorted_rid_marker_pos, d_num_common_markers[k], d_common_markers[k]);

    find_traceback <<<NUM_BLOCKS, BAND_SIZE>>>(n, maxSkip, d_marker_h[k], d_common_markers[k], d_num_common_markers[k], d_tb_mem[k], d_alignments[k], d_num_traceback[k]);


    err = hipMemcpy(h_num_traceback, d_num_traceback[k], n*sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "1. Error: hipMemcpy failed!\n");
        exit(1);
    }

    err = hipMemcpy(h_alignments, d_alignments[k], n*SHASTA_MAX_TB*sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "2. Error: hipMemcpy failed!\n");
        exit(1);
    }
    gettimeofday(&t3, NULL);
    
    if (report_time) {
        useconds = t2.tv_usec - t1.tv_usec;
        seconds = t2.tv_sec - t1.tv_sec;
        mseconds = ((seconds) * 1000 + useconds/1000.0) + 0.5;
        fprintf(stderr, "Time elapsed (t2-t1): %ld msec \n", mseconds);

        useconds = t3.tv_usec - t1.tv_usec;
        seconds = t3.tv_sec - t1.tv_sec;
        mseconds = ((seconds) * 1000 + useconds/1000.0) + 0.5;
        fprintf(stderr, "Time elapsed (t3-t1): %ld msec \n", mseconds);
    }

    gpu_lock[k].unlock();

    return;
}

extern "C" void shasta_shutdownProcessors(int nDevices) {
    for (int k=0; k<nDevices; k++) {
        hipFree(d_alignments[k]);

        hipFree(d_marker_h[k]);
        hipFree(d_tb_mem[k]);
        hipFree(d_num_traceback[k]);
        hipFree(d_common_markers[k]);
        hipFree(d_num_common_markers[k]);
    }
    free(d_alignments);

    free(d_marker_h);
    free(d_tb_mem);
    free(d_num_traceback);
    free(d_common_markers);
    free(d_num_common_markers);

    delete(gpu_lock);

}
